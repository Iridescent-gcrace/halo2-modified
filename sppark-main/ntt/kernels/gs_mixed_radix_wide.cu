#include "hip/hip_runtime.h"
// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

template <int intermediate_mul>
__launch_bounds__(768, 1) __global__
void _GS_NTT(const unsigned int radix, const unsigned int lg_domain_size,
             const unsigned int stage, const unsigned int iterations,
             fr_t* d_inout, const fr_t (*d_partial_twiddles)[WINDOW_SIZE],
             const fr_t* d_radix6_twiddles, const fr_t* d_radixX_twiddles,
             const fr_t* d_intermediate_twiddles,
             const unsigned int intermediate_twiddle_shift,
             const bool is_intt, const fr_t d_domain_size_inverse)
{
#if (__CUDACC_VER_MAJOR__-0) >= 11
    __builtin_assume(lg_domain_size <= MAX_LG_DOMAIN_SIZE);
    __builtin_assume(radix <= lg_domain_size);
    __builtin_assume(stage <= lg_domain_size);
#endif
    const index_t tid = threadIdx.x + blockDim.x * (index_t)blockIdx.x;

    const index_t inp_ntt_size = (index_t)1 << (stage - 1);
    //const index_t out_ntt_size = (index_t)1 << (stage - iterations - 1); // TODO: UNUSED

    // rearrange |tid|'s bits
    index_t idx0 = (tid & ~(inp_ntt_size - 1)) * 2;
    idx0 += (tid << (stage - iterations)) & (inp_ntt_size - 1);
    idx0 += tid >> (iterations - 1);
    idx0 -= ((tid >> (stage - 1)) << (stage - iterations));
    index_t idx1 = idx0 + inp_ntt_size;

    fr_t r0 = d_inout[idx0];
    fr_t r1 = d_inout[idx1];

    for (int s = iterations; --s >= 6;) {
        unsigned int laneMask = 1 << (s - 1);
        unsigned int thrdMask = (1 << s) - 1;
        unsigned int rank = threadIdx.x & thrdMask;

        fr_t t = d_radixX_twiddles[rank << (radix - (s + 1))];

        t *= (r0 - r1);
        r0 = r0 + r1;
        r1 = t;

        extern __shared__ fr_t shared_exchange[];
        extern __shared__ index_t shared_exchange_idx[];

        bool pos = rank < laneMask;
#ifdef __CUDA_ARCH__
        t = fr_t::csel(r1, r0, pos);
        __syncthreads();
        shared_exchange[threadIdx.x] = t;
        __syncthreads();
        t = shared_exchange[threadIdx.x ^ laneMask];
        r0 = fr_t::csel(t, r0, !pos);
        r1 = fr_t::csel(t, r1, pos);
#endif
        if (pos)
            swap(idx0, idx1);
        __syncthreads();
        shared_exchange_idx[threadIdx.x] = idx0;
        __syncthreads();
        idx0 = shared_exchange_idx[threadIdx.x ^ laneMask];
        if (pos)
            swap(idx0, idx1);
    }

    for (int s = min(iterations, 6); --s >= 1;) {
        unsigned int laneMask = 1 << (s - 1);
        unsigned int thrdMask = (1 << s) - 1;
        unsigned int rank = threadIdx.x & thrdMask;

        fr_t t = d_radix6_twiddles[rank << (6 - (s + 1))];

        t *= (r0 - r1);
        r0 = r0 + r1;
        r1 = t;

        bool pos = rank < laneMask;
#ifdef __CUDA_ARCH__
        t = fr_t::csel(r1, r0, pos);
        shfl_bfly(t, laneMask);
        r0 = fr_t::csel(t, r0, !pos);
        r1 = fr_t::csel(t, r1, pos);
#endif
        if (pos)
            swap(idx0, idx1);
        shfl_bfly(idx0, laneMask);
        if (pos)
            swap(idx0, idx1);
    }

    {
        fr_t t = r0 - r1;
        r0 = r0 + r1;
        r1 = t;
    }

    if (intermediate_mul == 1) {
        index_t thread_ntt_pos = (tid & (inp_ntt_size - 1)) >> (iterations - 1);
        unsigned int diff_mask = (1 << (iterations - 1)) - 1;
        unsigned int thread_ntt_idx = (tid & diff_mask) * 2;
        unsigned int nbits = MAX_LG_DOMAIN_SIZE - (stage - iterations);

        index_t root_idx0 = bit_rev(thread_ntt_idx, nbits) * thread_ntt_pos;
        index_t root_idx1 = thread_ntt_pos << (nbits - 1);

        fr_t first_root, second_root;
        get_intermediate_roots(first_root, second_root,
                               root_idx0, root_idx1, d_partial_twiddles);
        second_root *= first_root;

        r0 *= first_root;
        r1 *= second_root;
    } else if (intermediate_mul == 2) {
        index_t thread_ntt_pos = (tid & (inp_ntt_size - 1)) >> (iterations - 1);
        unsigned int diff_mask = (1 << (iterations - 1)) - 1;
        unsigned int thread_ntt_idx = (tid & diff_mask) * 2;
        unsigned int nbits = intermediate_twiddle_shift + iterations;

        index_t root_idx0 = bit_rev(thread_ntt_idx, nbits);
        index_t root_idx1 = bit_rev(thread_ntt_idx + 1, nbits);

        fr_t t0 = d_intermediate_twiddles[(thread_ntt_pos << radix) + root_idx0];
        fr_t t1 = d_intermediate_twiddles[(thread_ntt_pos << radix) + root_idx1];

        r0 *= t0;
        r1 *= t1;
    }

    if (is_intt && stage == iterations) {
        r0 *= d_domain_size_inverse;
        r1 *= d_domain_size_inverse;
    }

    d_inout[idx0] = r0;
    d_inout[idx1] = r1;
}

#define NTT_ARGUMENTS \
        unsigned int, unsigned int, unsigned int, unsigned int, fr_t*, \
        const fr_t (*)[WINDOW_SIZE], const fr_t*, const fr_t*, const fr_t*, \
        unsigned int, bool, fr_t

template __global__ void _GS_NTT<0>(NTT_ARGUMENTS);
template __global__ void _GS_NTT<1>(NTT_ARGUMENTS);
template __global__ void _GS_NTT<2>(NTT_ARGUMENTS);

#undef NTT_ARGUMENTS

#ifndef __CUDA_ARCH__

class GS_launcher {
    fr_t* d_inout;
    const int lg_domain_size;
    bool is_intt;
    int stage;
    const NTTParameters& ntt_parameters;
    const hipStream_t& stream;

public:
    GS_launcher(fr_t* d_ptr, int lg_dsz, bool innt,
                const NTTParameters& params, const hipStream_t& s)
      : d_inout(d_ptr), lg_domain_size(lg_dsz), is_intt(innt), stage(lg_dsz),
        ntt_parameters(params), stream(s)
    {}

    void step(int iterations)
    {
        assert(iterations <= 10);

        const int radix = iterations < 6 ? 6 : iterations;

        index_t num_threads = (index_t)1 << (lg_domain_size - 1);
        index_t block_size = 1 << (radix - 1);
        index_t num_blocks;

        block_size = (num_threads <= block_size) ? num_threads : block_size;
        num_blocks = (num_threads + block_size - 1) / block_size;

        assert(num_blocks == (unsigned int)num_blocks);

        fr_t* d_radixX_twiddles = nullptr;
        fr_t* d_intermediate_twiddles = nullptr;
        int intermediate_twiddle_shift = 0;

        #define NTT_CONFIGURATION \
            num_blocks, block_size, sizeof(fr_t) * block_size, stream

        #define NTT_ARGUMENTS \
            radix, lg_domain_size, stage, iterations, d_inout, \
            ntt_parameters.partial_twiddles, ntt_parameters.radix6_twiddles, \
            d_radixX_twiddles, d_intermediate_twiddles, \
            intermediate_twiddle_shift, \
            is_intt, domain_size_inverse[lg_domain_size]

        switch (radix) {
        case 6:
            switch (stage) {
            case 6:
                _GS_NTT<0><<<NTT_CONFIGURATION>>>(NTT_ARGUMENTS);
                break;
            case 12:
                intermediate_twiddle_shift = std::max(12 - lg_domain_size, 0);
                d_intermediate_twiddles = ntt_parameters.radix6_twiddles_6;
                _GS_NTT<2><<<NTT_CONFIGURATION>>>(NTT_ARGUMENTS);
                break;
            case 18:
                intermediate_twiddle_shift = std::max(18 - lg_domain_size, 0);
                d_intermediate_twiddles = ntt_parameters.radix6_twiddles_12;
                _GS_NTT<2><<<NTT_CONFIGURATION>>>(NTT_ARGUMENTS);
                break;
            default:
                _GS_NTT<1><<<NTT_CONFIGURATION>>>(NTT_ARGUMENTS);
                break;
            }
            break;
        case 7:
            d_radixX_twiddles = ntt_parameters.radix7_twiddles;
            switch (stage) {
            case 7:
                _GS_NTT<0><<<NTT_CONFIGURATION>>>(NTT_ARGUMENTS);
                break;
            case 14:
                intermediate_twiddle_shift = std::max(14 - lg_domain_size, 0);
                d_intermediate_twiddles = ntt_parameters.radix7_twiddles_7;
                _GS_NTT<2><<<NTT_CONFIGURATION>>>(NTT_ARGUMENTS);
                break;
            default:
                _GS_NTT<1><<<NTT_CONFIGURATION>>>(NTT_ARGUMENTS);
                break;
            }
            break;
        case 8:
            d_radixX_twiddles = ntt_parameters.radix8_twiddles;
            switch (stage) {
            case 8:
                _GS_NTT<0><<<NTT_CONFIGURATION>>>(NTT_ARGUMENTS);
                break;
            case 16:
                intermediate_twiddle_shift = std::max(16 - lg_domain_size, 0);
                d_intermediate_twiddles = ntt_parameters.radix8_twiddles_8;
                _GS_NTT<2><<<NTT_CONFIGURATION>>>(NTT_ARGUMENTS);
                break;
            default:
                _GS_NTT<1><<<NTT_CONFIGURATION>>>(NTT_ARGUMENTS);
                break;
            }
            break;
        case 9:
            d_radixX_twiddles = ntt_parameters.radix9_twiddles;
            switch (stage) {
            case 9:
                _GS_NTT<0><<<NTT_CONFIGURATION>>>(NTT_ARGUMENTS);
                break;
            case 18:
                intermediate_twiddle_shift = std::max(18 - lg_domain_size, 0);
                d_intermediate_twiddles = ntt_parameters.radix9_twiddles_9;
                _GS_NTT<2><<<NTT_CONFIGURATION>>>(NTT_ARGUMENTS);
                break;
            default:
                _GS_NTT<1><<<NTT_CONFIGURATION>>>(NTT_ARGUMENTS);
                break;
            }
            break;
        case 10:
            d_radixX_twiddles = ntt_parameters.radix10_twiddles;
            switch (stage) {
            case 10:
                _GS_NTT<0><<<NTT_CONFIGURATION>>>(NTT_ARGUMENTS);
                break;
            default:
                _GS_NTT<1><<<NTT_CONFIGURATION>>>(NTT_ARGUMENTS);
                break;
            }
            break;
        default:
            assert(false);
        }

        #undef NTT_CONFIGURATION
        #undef NTT_ARGUMENTS

        CUDA_OK(hipGetLastError());

        stage -= iterations;
    }
};

void GS_NTT(fr_t* d_inout, const int lg_domain_size, const bool is_intt,
    const NTTParameters& ntt_parameters, const hipStream_t& stream)
{
    GS_launcher params{d_inout, lg_domain_size, is_intt, ntt_parameters, stream};

    if (lg_domain_size <= 10) {
        params.step(lg_domain_size);
    } else if (lg_domain_size <= 12) {
        params.step(lg_domain_size - 6);
        params.step(6);
    } else if (lg_domain_size <= 18) {
        params.step(lg_domain_size / 2 + lg_domain_size % 2);
        params.step(lg_domain_size / 2);
    } else if (lg_domain_size <= 30) {
        int step = lg_domain_size / 3;
        int rem = lg_domain_size % 3;
        params.step(step + (rem > 0));
        params.step(step + (rem > 1));
        params.step(step);
    } else if (lg_domain_size <= 40) {
        int step = lg_domain_size / 4;
        int rem = lg_domain_size % 4;
        params.step(step + (rem > 0));
        params.step(step + (rem > 1));
        params.step(step + (rem > 2));
        params.step(step);
    } else {
        assert(false);
    }
}

#endif
