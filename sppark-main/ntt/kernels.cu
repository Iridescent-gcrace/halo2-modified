#include "hip/hip_runtime.h"
// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#ifndef __NTT_KERNELS_CU__
#define __NTT_KERNELS_CU__

#include <hip/hip_cooperative_groups.h>

__device__ __forceinline__
index_t bit_rev(index_t i, unsigned int nbits)
{
    if (sizeof(i) == 4 || nbits <= 32)
        return __brev(i) >> (8*sizeof(unsigned int) - nbits);
    else
        return __brevll(i) >> (8*sizeof(unsigned long long) - nbits);
}

__device__ __forceinline__
void shfl_bfly(fr_t& r, int laneMask)
{
#ifdef __CUDA_ARCH__
    #pragma unroll
    for (int iter = 0; iter < r.len(); iter++)
        r[iter] = __shfl_xor_sync(0xFFFFFFFF, r[iter], laneMask);
#endif
}

__device__ __forceinline__
void shfl_bfly(index_t& index, int laneMask)
{
    index = __shfl_xor_sync(0xFFFFFFFF, index, laneMask);
}

template<typename T>
__device__ __forceinline__
void swap(T& u1, T& u2)
{
    T temp = u1;
    u1 = u2;
    u2 = temp;
}

// Permutes the data in an array such that data[i] = data[bit_reverse(i)]
// and data[bit_reverse(i)] = data[i]
__launch_bounds__(1024) __global__
void bit_rev_permutation(fr_t* d_out, const fr_t *d_in, uint32_t lg_domain_size)
{
    index_t i = threadIdx.x + blockDim.x * (index_t)blockIdx.x;
    index_t r = bit_rev(i, lg_domain_size);

    if (i < r || (d_out != d_in && i == r)) {
        fr_t t0 = d_in[i];
        fr_t t1 = d_in[r];
        d_out[r] = t0;
        d_out[i] = t1;
    }
}

__launch_bounds__(1024) __global__
void bit_rev_permutation_aux(fr_t* out, const fr_t* in, uint32_t lg_domain_size)
{
    extern __shared__ fr_t exchange[];
    fr_t (*xchg)[8][8] = reinterpret_cast<decltype(xchg)>(exchange);

    index_t step = (index_t)1 << (lg_domain_size - 3);
    index_t group_idx = (threadIdx.x + blockDim.x * (index_t)blockIdx.x) >> 3;
    uint32_t brev_limit = lg_domain_size - 6;
    index_t brev_mask = ((index_t)1 << brev_limit) - 1;
    index_t group_idx_brev =
        (group_idx & ~brev_mask) | bit_rev(group_idx & brev_mask, brev_limit);
    uint32_t group_thread = threadIdx.x & 0x7;
    uint32_t group_thread_rev = bit_rev(group_thread, 3);
    uint32_t group_in_block_idx = threadIdx.x >> 3;

    #pragma unroll
    for (uint32_t i = 0; i < 8; i++) {
        xchg[group_in_block_idx][i][group_thread_rev] =
            in[group_idx * 8 + i * step + group_thread];
    }

    __syncwarp();

    #pragma unroll
    for (uint32_t i = 0; i < 8; i++) {
        out[group_idx_brev * 8 + i * step + group_thread] =
            xchg[group_in_block_idx][group_thread_rev][i];
    }
}

__device__ __forceinline__
fr_t get_intermediate_root(index_t pow, const fr_t (*roots)[WINDOW_SIZE],
                           unsigned int nbits = MAX_LG_DOMAIN_SIZE)
{
    unsigned int off = 0;

    fr_t root = roots[off][pow % WINDOW_SIZE];
    #pragma unroll 1
    while (pow >>= LG_WINDOW_SIZE)
        root *= roots[++off][pow % WINDOW_SIZE];

    return root;
}

__launch_bounds__(1024) __global__
void LDE_distribute_powers(fr_t* d_inout, uint32_t lg_blowup, bool bitrev,
                           const fr_t (*gen_powers)[WINDOW_SIZE])
{
    index_t idx = threadIdx.x + blockDim.x * (index_t)blockIdx.x;
    index_t pow = idx;
    fr_t r = d_inout[idx];

    if (bitrev) {
        size_t domain_size = gridDim.x * (size_t)blockDim.x;
        assert((domain_size & (domain_size-1)) == 0);
        uint32_t lg_domain_size = 63 - __clzll(domain_size);

        pow = bit_rev(idx, lg_domain_size);
    }

    r = r * get_intermediate_root(pow << lg_blowup, gen_powers);

    d_inout[idx] = r;
}

__launch_bounds__(1024) __global__
void LDE_spread_distribute_powers(fr_t* out, fr_t* in,
                                  const fr_t (*gen_powers)[WINDOW_SIZE],
                                  uint32_t lg_domain_size, uint32_t lg_blowup)
{
    extern __shared__ fr_t exchange[]; // block size

    assert(lg_domain_size + lg_blowup <= MAX_LG_DOMAIN_SIZE);

    size_t domain_size = (size_t)1 << lg_domain_size;
    uint32_t blowup = 1u << lg_blowup;
    uint32_t stride = gridDim.x * blockDim.x;

    assert(&out[domain_size * (blowup - 1)] == &in[0] &&
           (stride & (stride-1)) == 0);

    index_t idx0 = blockDim.x * blockIdx.x;
    uint32_t thread_pos = threadIdx.x & (blowup - 1);

#if 0
    index_t iters = domain_size / stride;
#else
    index_t iters = domain_size >> (31 - __clz(stride));
#endif
    index_t iterx = (blowup - 1) * (iters >> lg_blowup);

    for (index_t iter = 0; iter < iters; iter++) {
        index_t idx = idx0 + threadIdx.x;

        fr_t r = in[idx];

        // TODO: winterfell does not shift by lg_blowup - need to resolve
        // discrepency with Polygon
#ifdef HERMEZ
        index_t pow = bit_rev(idx, lg_domain_size + lg_blowup);
#else
        index_t pow = bit_rev(idx, lg_domain_size);
#endif

        r = r * get_intermediate_root(pow, gen_powers);

        __syncthreads();

        exchange[threadIdx.x] = r;

        if (iter >= iterx)
            cooperative_groups::this_grid().sync();
        else
            __syncthreads();

        r.zero();
        for (uint32_t i = 0; i < blowup; i++) {
            uint32_t offset = i * blockDim.x + threadIdx.x;

            if (thread_pos == 0)
                r = exchange[offset >> lg_blowup];

            out[(idx0 << lg_blowup) + offset] = r;
        }

        idx0 += stride;
    }
}

__device__ __forceinline__
void get_intermediate_roots(fr_t& root0, fr_t& root1,
                            index_t idx0, index_t idx1,
                            const fr_t (*roots)[WINDOW_SIZE])
{
    int win = (WINDOW_NUM - 1) * LG_WINDOW_SIZE;
    int off = (WINDOW_NUM - 1);

    root0 = roots[off][idx0 >> win];
    root1 = roots[off][idx1 >> win];
    #pragma unroll 1
    while (off--) {
        win -= LG_WINDOW_SIZE;
        root0 *= roots[off][(idx0 >> win) % WINDOW_SIZE];
        root1 *= roots[off][(idx1 >> win) % WINDOW_SIZE];
    }
}

# include "kernels/gs_mixed_radix_wide.cu"
# include "kernels/ct_mixed_radix_wide.cu"

#endif /* __NTT_KERNELS_CU__ */
