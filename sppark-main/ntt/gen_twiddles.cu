#include "hip/hip_runtime.h"
// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#define ONE fr_t::one()

__global__
void generate_partial_twiddles(fr_t (*roots)[WINDOW_SIZE],
                               const fr_t root_of_unity)
{
    const unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    assert(tid < WINDOW_SIZE);
    fr_t root;

    if (tid == 0)
        root = ONE;
    else if (tid == 1)
        root = root_of_unity;
    else
        root = root_of_unity^tid;

    roots[0][tid] = root;

    for (int off = 1; off < WINDOW_NUM; off++) {
        for (int i = 0; i < LG_WINDOW_SIZE; i++)
#if defined(__CUDA_ARCH__)
            root.sqr();
#else
            root *= root;
#endif
        roots[off][tid] = root;
    }
}

__global__
void generate_all_twiddles(fr_t* d_radixX_twiddles, const fr_t root6,
                                                    const fr_t root7,
                                                    const fr_t root8,
                                                    const fr_t root9,
                                                    const fr_t root10)
{
    const unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int pow;
    fr_t root_of_unity;

    if (tid < 64) {
        pow = tid;
        root_of_unity = root7;
    } else if (tid < 64 + 128) {
        pow = tid - 64;
        root_of_unity = root8;
    } else if (tid < 64 + 128 + 256) {
        pow = tid - 64 - 128;
        root_of_unity = root9;
    } else if (tid < 64 + 128 + 256 + 512) {
        pow = tid - 64 - 128 - 256;
        root_of_unity = root10;
    } else if (tid < 64 + 128 + 256 + 512 + 32) {
        pow = tid - 64 - 128 - 256 - 512;
        root_of_unity = root6;
    } else {
        assert(false);
    }

    if (pow == 0)
        d_radixX_twiddles[tid] = ONE;
    else if (pow == 1)
        d_radixX_twiddles[tid] = root_of_unity;
    else
        d_radixX_twiddles[tid] = root_of_unity^pow;
}

__launch_bounds__(512) __global__
void generate_radixX_twiddles_X(fr_t* d_radixX_twiddles_X, int n,
                                const fr_t root_of_unity)
{
    if (gridDim.x == 1) {
        fr_t root0;

        d_radixX_twiddles_X[threadIdx.x] = ONE;
        d_radixX_twiddles_X += blockDim.x;

        if (threadIdx.x == 0)
            root0 = ONE;
        else if (threadIdx.x == 1)
            root0 = root_of_unity;
        else
            root0 = root_of_unity^threadIdx.x;

        d_radixX_twiddles_X[threadIdx.x] = root0;
        d_radixX_twiddles_X += blockDim.x;

        fr_t root1 = root0;

        for (int i = 2; i < n; i++) {
            root1 *= root0;
            d_radixX_twiddles_X[threadIdx.x] = root1;
            d_radixX_twiddles_X += blockDim.x;
        }
    } else {
        fr_t root0;

        if (threadIdx.x == 0)
            root0 = ONE;
        else
            root0 = root_of_unity ^ (threadIdx.x * gridDim.x);

        unsigned int pow = blockIdx.x * threadIdx.x;
        unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
        fr_t root1;

        if (pow == 0)
            root1 = ONE;
        else if (pow == 1)
            root1 = root_of_unity;
        else
            root1 = root_of_unity^pow;

        d_radixX_twiddles_X[tid] = root1;
        d_radixX_twiddles_X += gridDim.x * blockDim.x;

        for (int i = gridDim.x; i < n; i += gridDim.x) {
            root1 *= root0;
            d_radixX_twiddles_X[tid] = root1;
            d_radixX_twiddles_X += gridDim.x * blockDim.x;
        }
    }
}

#undef ONE
