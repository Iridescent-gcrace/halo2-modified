// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#if defined(FEATURE_BLS12_381)
# include <ff/bls12-381.hpp>
#elif defined(FEATURE_BLS12_377)
# include <ff/bls12-377.hpp>
#else
# error "no FEATURE"
#endif

#include <ntt/ntt.cuh>

#ifndef __CUDA_ARCH__

extern "C"
RustError compute_ntt(size_t device_id, fr_t* inout, uint32_t lg_domain_size,
                      NTT::InputOutputOrder ntt_order,
                      NTT::Direction ntt_direction,
                      NTT::Type ntt_type)
{
    auto& gpu = select_gpu(device_id);

    return NTT::Base(gpu, inout, lg_domain_size,
                     ntt_order, ntt_direction, ntt_type);
}

#endif
